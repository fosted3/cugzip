#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <cstdint>
#include <vector>
#include <cstdio>
#include <cassert>
#include "aux.h"
#include <thrust/sort.h>
#include <thrust/functional.h>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/random.h>
#include <thrust/inner_product.h>
#include <thrust/binary_search.h>
#include <thrust/adjacent_difference.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>

#define MATCH_LENGTH 1024

#define handle_error(ans) { cuda_assert((ans), __FILE__, __LINE__); }
inline void cuda_assert(hipError_t code, const char *file, int line, bool abort=true)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

void print_vector(const std::string &name, const thrust::device_vector<uint32_t> &v)
{
	std::cout << "  " << std::setw(20) << name << "  ";
	thrust::copy(v.begin(), v.end(), std::ostream_iterator<uint32_t>(std::cout, " "));
	std::cout << std::endl;
}

/*__global__ void lz77_find_matches(uint8_t *data, uint32_t size, uint32_t **matches, uint32_t *num_matches)
{
	uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t match_idx;
	if (idx < size - 2)
	{
		match_idx = *((uint32_t*) (data + idx)) & 0x00FFFFFF;
		if (*(matches[match_idx]) == 0)
		{
			atomicAdd(num_matches, 1);
			matches[match_idx] = (uint32_t*) malloc(MATCH_LENGTH * sizeof(uint32_t));
			matches[match_idx][0] = 1;
			matches[match_idx][1] = idx;
		}
		else
		{
			matches[match_idx][0]++;
			matches[match_idx][matches[match_idx][0]] = idx;
		}
	}
}*/

__global__ void lz77_stage1(uint8_t *data, uint32_t size, uint32_t *hashes, uint32_t *idx_list) //size is two less than sizeof(data)
{
	const uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
	{
		hashes[idx] = ((uint32_t) (data[idx + 2] << 16) | ((uint32_t) (data[idx + 1] << 8)) | ((uint32_t) (data[idx]))) & 0x00FFFFFF;
		//printf("Set idx %d data %x %x %x to %x\n", idx, data[idx], data[idx+1], data[idx+2], hashes[idx]);
		idx_list[idx] = idx;
		//printf("Set idx %d to %u\n", idx, idx_list[idx]);
	}
}

__global__ void lz77_stage2(uint32_t *sorted_hashes, uint32_t size, uint32_t *idx_list, uint32_t *out)
{
	const uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
	{
		if (sorted_hashes[idx] != sorted_hashes[idx + 1])
		{
			out[idx + 1] = idx_list[idx + 1];
		}
		else
		{
			out[idx + 1] = 0xFFFFFFFF;
		}
	}
}

lz77_data* lz77_cuda(std::vector<uint8_t> *data)
{
	uint32_t block_size = 1024;
	uint32_t grid_size = data -> size() / block_size;
	uint32_t data_size = data -> size() - 2;
	uint8_t *device_file = NULL;
	
	uint32_t *device_idx_list = NULL;
	uint32_t *device_hashes = NULL;
	//thrust::device_vector<uint32_t> device_hashes_vector;
	//uint32_t *device_hashes = thrust::raw_pointer_cast(device_hashes_vector.data());
	if (data -> size() % block_size) { grid_size++; }
	handle_error(hipMalloc(&device_file, data -> size()));
	handle_error(hipMalloc(&device_idx_list, data_size * sizeof(uint32_t)));
	handle_error(hipMalloc(&device_hashes, data_size * sizeof(uint32_t)));
	assert(device_file != NULL);
	assert(device_idx_list != NULL);
	assert(device_hashes != NULL);
	handle_error(hipMemcpy(device_file, data -> data(), data -> size(), hipMemcpyHostToDevice));
	lz77_stage1<<<grid_size, block_size>>>(device_file, data -> size() - 2, device_hashes, device_idx_list);
	handle_error(hipDeviceSynchronize());
	//thrust::stable_sort_by_key(device_hashes, device_hashes + data_size, device_idx_list); 
	thrust::device_vector<uint32_t> histogram_input(data_size, 0);
	thrust::device_vector<uint32_t> histogram_values;
	thrust::device_vector<uint32_t> histogram_counts;
	//thrust::device_ptr<uint32_t> hash_ptr(device_hashes);
	//thrust::copy(device_hashes, device_hashes + data_size, histogram_input.begin());
	//thrust::copy(device_hashes_vector.begin(), device_hashes_vector.end(), histogram_input.begin());
	handle_error(hipMemcpy(thrust::raw_pointer_cast(histogram_input.data()), device_hashes, data_size * sizeof(uint32_t), hipMemcpyDeviceToDevice));
	thrust::sort(histogram_input.begin(), histogram_input.end());
	uint32_t unique_keys = thrust::inner_product(histogram_input.begin(), histogram_input.end() - 1, histogram_input.begin() + 1, (uint32_t) 1, thrust::plus<uint32_t>(), thrust::not_equal_to<uint32_t>());
	histogram_values.resize(unique_keys);
	histogram_counts.resize(unique_keys);
	thrust::reduce_by_key(histogram_input.begin(), histogram_input.end(), thrust::constant_iterator<uint32_t>(1), histogram_values.begin(), histogram_counts.begin());
	print_vector("histogram values", histogram_values);
	print_vector("histogram counts", histogram_counts);
	return NULL;
}
